#include "hip/hip_runtime.h"
#include <iostream>
#include "oddsEnds.h"


__global__ void sequence_gpu(int *d_ptr, int length)
{
    int elemID = blockIdx.x * blockDim.x + threadIdx.x;

    if (elemID < length)
    {
        d_ptr[elemID] = elemID;
    }
}



int default_test()
{
    using namespace std;
    
    cout << "CUDA Runtime API template" << endl;
    cout << "=========================" << endl;
    cout << "Self-test started" << endl;

    const int N = 100;

    int *d_ptr;
    ASSERT(hipSuccess == hipMalloc(&d_ptr, N * sizeof(int)), "Device allocation of " << N << " ints failed", -1);

    int *h_ptr;
    ASSERT(hipSuccess == hipHostMalloc(&h_ptr, N * sizeof(int)), "Host allocation of "   << N << " ints failed", -1);

    cout << "Memory allocated successfully" << endl;

    dim3 cudaBlockSize(32,1,1);
    dim3 cudaGridSize((N + cudaBlockSize.x - 1) / cudaBlockSize.x, 1, 1);
    sequence_gpu<<<cudaGridSize, cudaBlockSize>>>(d_ptr, N);
    ASSERT(hipSuccess == hipGetLastError(), "Kernel launch failed", -1);
    ASSERT(hipSuccess == hipDeviceSynchronize(), "Kernel synchronization failed", -1);

    sequence_cpu(h_ptr, N);

    cout << "CUDA and CPU algorithm implementations finished" << endl;

    int *h_d_ptr;
    ASSERT(hipSuccess == hipHostMalloc(&h_d_ptr, N *sizeof(int)), "Host allocation of " << N << " ints failed", -1);
    ASSERT(hipSuccess == hipMemcpy(h_d_ptr, d_ptr, N *sizeof(int), hipMemcpyDeviceToHost), "Copy of " << N << " ints from device to host failed", -1);
    bool bValid = true;

    for (int i=0; i<N && bValid; i++)
    {
        if (h_ptr[i] != h_d_ptr[i])
        {
            bValid = false;
        }
    }

    ASSERT(hipSuccess == hipFree(d_ptr),       "Device deallocation failed", -1);
    ASSERT(hipSuccess == hipHostFree(h_ptr),   "Host deallocation failed",   -1);
    ASSERT(hipSuccess == hipHostFree(h_d_ptr), "Host deallocation failed",   -1);

    cout << "Memory deallocated successfully" << endl;
    cout << "TEST Results " << endl;

    return bValid;
}